// pinned memory(cudaMallocHost,cudaMalloc,cudaMemcpy)
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include<cassert>
#include <iostream>
using namespace std;

__global__ void vector_add(int *a,int *b,int *c,int N){
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    if(tid<N){
    c[tid]=a[tid]+b[tid];
    }
}

void init_array(int *a, int N){
    for(int i=0;i<N;i++){
        a[i]=rand()%100;
    }
}

void checker(int *a,int *b,int *c, int N){
    for(int i=0;i<N;i++){
        assert(a[i]+b[i]==c[i]);
    }
    cout<<"Program completed!";
}

int main(){
    int N=1<<20; //2^20 1048576
    size_t bytes=N*sizeof(bytes);

    int *a,*b,*c;
    hipHostMalloc(&a,bytes, hipHostMallocDefault);
    hipHostMalloc(&b,bytes, hipHostMallocDefault);
    hipHostMalloc(&c,bytes, hipHostMallocDefault);

    init_array(a,N);
    init_array(b,N);

    int *da,*db,*dc;
    hipMalloc(&da,bytes);
    hipMalloc(&db,bytes);
    hipMalloc(&dc,bytes);

    hipMemcpy(da,a,bytes,hipMemcpyHostToDevice);
    hipMemcpy(db,b,bytes,hipMemcpyHostToDevice);

    // CTA and grid dimensions
    int threads=256;
    int blocks=(N+threads-1)/threads;

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vector_add<<<blocks,threads>>>(da,db,dc,N);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds=0;
    hipEventElapsedTime(&milliseconds,start,stop);
    cout<<"The total time taken by the add kernel is: "<<milliseconds<<" ms"<<endl;
    hipMemcpy(c,dc,bytes,hipMemcpyDeviceToHost);

    checker(a,b,c,N);
}
